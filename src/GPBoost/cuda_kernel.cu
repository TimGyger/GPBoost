
#include <hip/hip_runtime.h>
/*!
* This file is part of GPBoost a C++ library for combining
*	boosting with Gaussian process and mixed effects models
*
* Copyright (c) 2020 Fabio Sigrist. All rights reserved.
*
* Licensed under the Apache License Version 2.0. See LICENSE file in the project root for license information.
*/
#ifdef USE_CUDA_GP

#include <GPBoost/GP_utils.h>
#include <cuda_runtime.h>
#include <cublas_v2.h>
#include <cusparse.h>
#include <device_launch_parameters.h>
#include <cusolverDn.h>
#include <LightGBM/utils/log.h>
using LightGBM::Log;

namespace GPBoost {

    bool try_matmul_gpu(const den_mat_t& A, const den_mat_t& B, den_mat_t& C) {
        int M = A.rows(), K = A.cols(), N = B.cols();
        if (K != B.rows()) {
            Log::REInfo("[GPU] Dimension mismatch.");
            return false;
        }

        C.resize(M, N);

        const double* h_A = A.data();
        const double* h_B = B.data();
        double* h_C = C.data();

        double* d_A = nullptr, * d_B = nullptr, * d_C = nullptr;
        cudaError_t cuda_stat;
        cublasStatus_t stat;
        cublasHandle_t handle;

        size_t size_A = M * K * sizeof(double);
        size_t size_B = K * N * sizeof(double);
        size_t size_C = M * N * sizeof(double);

        cuda_stat = cudaMalloc((void**)&d_A, size_A);
        if (cuda_stat != cudaSuccess) return false;
        cuda_stat = cudaMalloc((void**)&d_B, size_B);
        if (cuda_stat != cudaSuccess) {
            cudaFree(d_A);
            return false;
        }

        cuda_stat = cudaMalloc((void**)&d_C, size_C);
        if (cuda_stat != cudaSuccess) {
            cudaFree(d_A); cudaFree(d_B);
            return false;
        }

        cudaMemcpy(d_A, h_A, size_A, cudaMemcpyHostToDevice);
        cudaMemcpy(d_B, h_B, size_B, cudaMemcpyHostToDevice);

        stat = cublasCreate(&handle);
        if (stat != CUBLAS_STATUS_SUCCESS) {
            cudaFree(d_A); cudaFree(d_B); cudaFree(d_C);
            return false;
        }

        const double alpha = 1.0;
        const double beta = 0.0;

        // cuBLAS performs: C = alpha * op(A) * op(B) + beta * C
        // We want: C = A * B
        // A: MxK, B: KxN, C: MxN
        // So op(A) = A, op(B) = B
        stat = cublasDgemm(handle,
            CUBLAS_OP_N, CUBLAS_OP_N,  // No transpose
            M, N, K,                   // C is MxN, A is MxK, B is KxN
            &alpha,
            d_A, M,  // lda = leading dim of A = M (since column-major)
            d_B, K,  // ldb = leading dim of B = K
            &beta,
            d_C, M); // ldc = leading dim of C = M

        if (stat != CUBLAS_STATUS_SUCCESS) {
            cublasDestroy(handle);
            cudaFree(d_A); cudaFree(d_B); cudaFree(d_C);
            return false;
        }

        cudaMemcpy(h_C, d_C, size_C, cudaMemcpyDeviceToHost);

        cublasDestroy(handle);
        cudaFree(d_A); cudaFree(d_B); cudaFree(d_C);

        Log::REInfo("[GPU] Matrix multiplication completed with cuBLAS.");
        return true;
    }

    bool try_diag_times_dense_gpu(const vec_t& D, const den_mat_t& B, den_mat_t& C) {
        int M = B.rows();
        int N = B.cols();

        if (D.size() != M) {
            Log::REInfo("[GPU] Dimension mismatch between diagonal and matrix.");
            return false;
        }

        C.resize(M, N);

        // Host pointers
        const double* h_D = D.data();
        const double* h_B = B.data();
        double* h_C = C.data();

        // Device pointers
        double* d_D = nullptr;
        double* d_B = nullptr;
        double* d_C = nullptr;

        cudaMalloc((void**)&d_D, M * sizeof(double));
        cudaMalloc((void**)&d_B, M * N * sizeof(double));
        cudaMalloc((void**)&d_C, M * N * sizeof(double));

        cudaMemcpy(d_D, h_D, M * sizeof(double), cudaMemcpyHostToDevice);
        cudaMemcpy(d_B, h_B, M * N * sizeof(double), cudaMemcpyHostToDevice);
        // Create cuBLAS handle
        cublasHandle_t handle;
        cublasCreate(&handle);
        // Multiply: C = diag(D) * B (i.e., scale each row of B by D[i])
        // Use cuBLAS: d_C = diag(d_D) * d_B
        cublasStatus_t stat = cublasDdgmm(handle,
            CUBLAS_SIDE_LEFT, // Left = scale rows (use RIGHT to scale columns)
            M, N,
            d_B, M,
            d_D, 1, // stride = 1
            d_C, M);
        if (stat != CUBLAS_STATUS_SUCCESS) {
            Log::REInfo("[GPU] cuBLAS Ddgmm failed.");
            cudaFree(d_D); cudaFree(d_B); cudaFree(d_C);
            cublasDestroy(handle);
            return false;
        }

        cudaMemcpy(h_C, d_C, M * N * sizeof(double), cudaMemcpyDeviceToHost);

        // Clean up
        cudaFree(d_D);
        cudaFree(d_B);
        cudaFree(d_C);
        cublasDestroy(handle);

        Log::REInfo("[GPU] Diagonal x Dense matrix multiplication completed with cuBLAS.");
        return true;
    }

    bool try_sparse_dense_matmul_gpu(const sp_mat_rm_t& A, const den_mat_t& B, den_mat_t& C) {
        int M = A.rows(), K = A.cols(), N = B.cols();
        if (K != B.rows()) {
            Log::REInfo("[GPU] Dimension mismatch.");
            return false;
        }

        //C.resize(M, N);

        // Convert Eigen sparse matrix to CSR format (cuSPARSE prefers CSR)
        const int nnz = A.nonZeros();
        const int* h_csrOffsets = A.outerIndexPtr();  // Row pointers
        const int* h_columns = A.innerIndexPtr();     // Column indices
        const double* h_values = A.valuePtr();        // Non-zero values

        // Allocate device memory
        int* d_csrOffsets;
        int* d_columns;
        double* d_values;
        double* d_B;
        double* d_C;

        cudaMalloc((void**)&d_csrOffsets, (M + 1) * sizeof(int));
        cudaMalloc((void**)&d_columns, nnz * sizeof(int));
        cudaMalloc((void**)&d_values, nnz * sizeof(double));
        cudaMalloc((void**)&d_B, K * N * sizeof(double));
        cudaMalloc((void**)&d_C, M * N * sizeof(double));

        cudaMemcpy(d_csrOffsets, h_csrOffsets, (M + 1) * sizeof(int), cudaMemcpyHostToDevice);
        cudaMemcpy(d_columns, h_columns, nnz * sizeof(int), cudaMemcpyHostToDevice);
        cudaMemcpy(d_values, h_values, nnz * sizeof(double), cudaMemcpyHostToDevice);
        cudaMemcpy(d_B, B.data(), K * N * sizeof(double), cudaMemcpyHostToDevice);
        //cudaMemset(d_C, 0, M * N * sizeof(double));
        
        // Create cuSPARSE handle and descriptors
        cusparseHandle_t handle;
        cusparseCreate(&handle);
        
        cusparseSpMatDescr_t matA;
        cusparseDnMatDescr_t matB, matC;
        cusparseCreateCsr(&matA, M, K, nnz,
            d_csrOffsets, d_columns, d_values,
            CUSPARSE_INDEX_32I, CUSPARSE_INDEX_32I,
            CUSPARSE_INDEX_BASE_ZERO, CUDA_R_64F);
       
        cusparseCreateDnMat(&matB, K, N, K, d_B, CUDA_R_64F, CUSPARSE_ORDER_COL);
        cusparseCreateDnMat(&matC, M, N, M, d_C, CUDA_R_64F, CUSPARSE_ORDER_COL);
        
        const double alpha = 1.0;
        const double beta = 0.0;

        size_t bufferSize = 0;
        void* dBuffer = nullptr;
        cusparseSpMM_bufferSize(handle,
            CUSPARSE_OPERATION_NON_TRANSPOSE,
            CUSPARSE_OPERATION_NON_TRANSPOSE,
            &alpha, matA, matB, &beta, matC,
            CUDA_R_64F, CUSPARSE_SPMM_CSR_ALG2,
            &bufferSize);
        
        cusparseStatus_t stat = cusparseSpMM(handle,
            CUSPARSE_OPERATION_NON_TRANSPOSE,
            CUSPARSE_OPERATION_NON_TRANSPOSE,
            &alpha, matA, matB, &beta, matC,
            CUDA_R_64F, CUSPARSE_SPMM_CSR_ALG2,
            dBuffer);
        
        if (stat != CUSPARSE_STATUS_SUCCESS) {
            Log::REInfo("[GPU] cuSPARSE SpMM failed.");
            cusparseDestroySpMat(matA);
            cusparseDestroyDnMat(matB);
            cusparseDestroyDnMat(matC);
            cusparseDestroy(handle);
            cudaFree(dBuffer); cudaFree(d_csrOffsets); cudaFree(d_columns);
            cudaFree(d_values); cudaFree(d_B); cudaFree(d_C);
            return false;
        }

        // Copy result back to host
        cudaMemcpy(C.data(), d_C, M * N * sizeof(double), cudaMemcpyDeviceToHost);

        // Clean up
        cusparseDestroySpMat(matA);
        cusparseDestroyDnMat(matB);
        cusparseDestroyDnMat(matC);
        cusparseDestroy(handle);

        cudaFree(dBuffer);
        cudaFree(d_csrOffsets);
        cudaFree(d_columns);
        cudaFree(d_values);
        cudaFree(d_B);
        cudaFree(d_C);
        
        Log::REInfo("[GPU] Sparse x Dense matrix multiplication completed with cuSPARSE.");
        return true;
    }

    bool try_solve_lower_triangular_gpu(const chol_den_mat_t& chol, const den_mat_t& R_host, den_mat_t& X_host) {
        den_mat_t L_host = chol.matrixL();
        int n = L_host.rows();
        int m = R_host.cols();
        if (L_host.cols() != n || R_host.rows() != n) {
            return false;
        }
        X_host.resize(n, m);
        // Allocate device memory
        double* d_L = nullptr;
        double* d_X = nullptr;

        cudaMalloc(&d_L, n * n * sizeof(double));
        cudaMalloc(&d_X, n * m * sizeof(double));

        cudaMemcpy(d_L, L_host.data(), n * n * sizeof(double), cudaMemcpyHostToDevice);
        cudaMemcpy(d_X, R_host.data(), n * m * sizeof(double), cudaMemcpyHostToDevice);

        // Create cuBLAS handle
        cublasHandle_t handle;
        cublasStatus_t stat = cublasCreate(&handle);
        if (stat != CUBLAS_STATUS_SUCCESS) {
            cudaFree(d_L); cudaFree(d_X);
            return false;
        }
        const double alpha = 1.0;

        // Solve: L * X = R -> X = L^{-1} * R
        // L is lower-triangular, column-major
        // Left-side, lower-triangular, no transpose, non-unit diagonal
        stat = cublasDtrsm(
            handle,
            CUBLAS_SIDE_LEFT,      // Solve L * X = R
            CUBLAS_FILL_MODE_LOWER,
            CUBLAS_OP_N,           // No transpose
            CUBLAS_DIAG_NON_UNIT,  // Assume general diagonal
            n,                     // number of rows of L and X
            m,                     // number of columns of X
            &alpha,                // Scalar alpha
            d_L, n,                // L, leading dimension n
            d_X, n                 // R becomes X, leading dimension n
        );

        if (stat != CUBLAS_STATUS_SUCCESS) {
            cudaFree(d_L); cudaFree(d_X);
            cublasDestroy(handle);
            return false;
        }

        // Copy result back
        cudaMemcpy(X_host.data(), d_X, n * m * sizeof(double), cudaMemcpyDeviceToHost);

        // Cleanup
        cudaFree(d_L);
        cudaFree(d_X);
        cublasDestroy(handle);

        Log::REInfo("[GPU] Triangular solve with CUBLAS.");
        return true;
    }

    

    // CUDA kernel: Sigma(i,j) -= dot(M1.col(i), M2.col(j))
    __global__ void subtract_prod_from_mat_kernel(
        const double* __restrict__ M1,
        const double* __restrict__ M2,
        double* Sigma,
        int M1_rows, int M1_cols,
        int M2_rows, int M2_cols,
        bool only_triangular)
    {
        int i = blockIdx.y * blockDim.y + threadIdx.y;
        int j = blockIdx.x * blockDim.x + threadIdx.x;

        if (i >= M1_cols || j >= M2_cols) return;
        if (only_triangular && j < i) return;

        double dot = 0.0;
        for (int k = 0; k < M1_rows; ++k) {
            dot += M1[i * M1_rows + k] * M2[j * M2_rows + k];
        }

        // column-major access: Sigma(i, j) => j * rows + i
        atomicAdd(&Sigma[j * M1_cols + i], -dot);

        if (!only_triangular && j > i) {
            atomicAdd(&Sigma[i * M1_cols + j], -dot);  // symmetric fill
        }
    }
    __global__ void subtract_prod_from_sparse_mat_kernel(
    const int* __restrict__ row_ptr,
    const int* __restrict__ col_idx,
    double* __restrict__ values,
    const double* __restrict__ M1,  // Shape: (n_rows, K)
    const double* __restrict__ M2,  // Shape: (n_cols, K)
    int n_rows, int n_cols, int K)
{
    int row = blockIdx.x * blockDim.x + threadIdx.x;
    if (row >= n_rows) return;

    int row_start = row_ptr[row];
    int row_end = row_ptr[row + 1];

    for (int idx = row_start; idx < row_end; ++idx) {
        int col = col_idx[idx];

        // Only compute upper triangle or diagonal
        if (row <= col) {
            double dot = 0.0;
            for (int k = 0; k < K; ++k) {
                dot += M1[row * K + k] * M2[col * K + k];
            }
            atomicAdd(&values[idx], -dot);
        }
            // Note: for full symmetry, the host must mirror Sigma(j,i) = Sigma(i,j) afterwards
    }
}

    void launch_subtract_sparse_kernel(
        const int* row_ptr, const int* col_idx, double* values,
        const double* M1, const double* M2,
        int n, int m, int K, bool only_triangular)
    {
        int blockSize = 256;
        int numBlocks = (n + blockSize - 1) / blockSize;
        subtract_prod_from_sparse_mat_kernel << <numBlocks, blockSize >> > (
            row_ptr, col_idx, values, M1, M2, n, m, K);
    }

    void launch_subtract_prod_from_mat_kernel(
        const double* M1, const double* M2, double* Sigma,
        int M1_rows, int M1_cols,
        int M2_rows, int M2_cols,
        bool only_triangular)
    {
        dim3 blockDim(16, 16);
        dim3 gridDim((M2_cols + blockDim.x - 1) / blockDim.x,
            (M1_cols + blockDim.y - 1) / blockDim.y);

        subtract_prod_from_mat_kernel << <gridDim, blockDim >> > (
            M1, M2, Sigma,
            M1_rows, M1_cols,
            M2_rows, M2_cols,
            only_triangular
            );
        cudaDeviceSynchronize();
    }

    
    bool cholesky_cusolver_to_eigen(chol_den_mat_t& llt, const den_mat_t& A_input) {
        int N = A_input.rows();
        if (A_input.cols() != N) {
            Log::REInfo("Input matrix is not square.");
            return false;
        }

        // Step 1: Create cuSolver handle
        cusolverDnHandle_t handle;
        cusolverStatus_t status = cusolverDnCreate(&handle);
        if (status != CUSOLVER_STATUS_SUCCESS) {
            Log::REInfo("cuSOLVER initialization failed.");
            return false;
        }

        // Step 2: Allocate GPU memory for matrix
        double* d_A = nullptr;
        cudaError_t cudaStat = cudaMalloc(&d_A, sizeof(double) * N * N);
        if (cudaStat != cudaSuccess) {
            Log::REInfo("cudaMalloc failed for d_A");
            cusolverDnDestroy(handle);
            return false;
        }

        cudaStat = cudaMemcpy(d_A, A_input.data(), sizeof(double) * N * N, cudaMemcpyHostToDevice);
        if (cudaStat != cudaSuccess) {
            Log::REInfo("cudaMemcpy failed");
            cudaFree(d_A);
            cusolverDnDestroy(handle);
            return false;
        }

        // Step 3: Query buffer size
        int work_size = 0;
        status = cusolverDnDpotrf_bufferSize(handle, CUBLAS_FILL_MODE_LOWER, N, d_A, N, &work_size);
        if (status != CUSOLVER_STATUS_SUCCESS) {
            Log::REInfo("cusolverDnDpotrf_bufferSize failed.");
            cudaFree(d_A);
            cusolverDnDestroy(handle);
            return false;
        }

        double* work = nullptr;
        cudaStat = cudaMalloc(&work, sizeof(double) * work_size);
        if (cudaStat != cudaSuccess) {
            Log::REInfo("cudaMalloc failed for workspace");
            cudaFree(d_A);
            cusolverDnDestroy(handle);
            return false;
        }

        int* dev_info = nullptr;
        cudaStat = cudaMalloc(&dev_info, sizeof(int));
        if (cudaStat != cudaSuccess) {
            Log::REInfo("cudaMalloc failed ");
            cudaFree(d_A);
            cudaFree(work);
            cusolverDnDestroy(handle);
            return false;
        }

        // Step 4: Compute Cholesky factorization
        status = cusolverDnDpotrf(handle, CUBLAS_FILL_MODE_LOWER, N, d_A, N, work, work_size, dev_info);
        if (status != CUSOLVER_STATUS_SUCCESS) {
            Log::REInfo("cusolverDnDpotrf failed.");
            cudaFree(d_A); cudaFree(work); cudaFree(dev_info);
            cusolverDnDestroy(handle);
            return false;
        }

        int dev_info_h = 0;
        cudaStat = cudaMemcpy(&dev_info_h, dev_info, sizeof(int), cudaMemcpyDeviceToHost);
        if (cudaStat != cudaSuccess) {
            Log::REInfo("cudaMemcpy failed");
            cudaFree(d_A); cudaFree(work); cudaFree(dev_info);
            cusolverDnDestroy(handle);
            return false;
        }

        if (dev_info_h != 0) {
            Log::REInfo("Cholesky factorization failed on GPU");
            cudaFree(d_A); cudaFree(work); cudaFree(dev_info);
            cusolverDnDestroy(handle);
            return false;
        }

        // Step 5: Copy result back (only lower triangle)
        den_mat_t L(N, N);
        cudaStat = cudaMemcpy(L.data(), d_A, sizeof(double) * N * N, cudaMemcpyDeviceToHost);
        if (cudaStat != cudaSuccess) {
            Log::REInfo("cudaMemcpy failed");
            cudaFree(d_A); cudaFree(work); cudaFree(dev_info);
            cusolverDnDestroy(handle);
            return false;
        }

        // Step 6: Feed to Eigen's LLT (only lower triangle will be used)
        llt.compute(L.selfadjointView<Eigen::Lower>());

        // Step 7: Cleanup
        cudaFree(d_A);
        cudaFree(work);
        cudaFree(dev_info);
        cusolverDnDestroy(handle);

        Log::REInfo("[GPU] Cholesky factorization with cuSOLVER completed successfully.");
        return true;
    }

}  // namespace GPBoost

#endif  // USE_CUDA_GP
